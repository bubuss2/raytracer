#include "hip/hip_runtime.h"
#include <iostream>
#include <time.h>

#include "ray.h"
#include "vector3.h"

// limited version of checkCudaErrors from hip/hip_runtime_api.h in CUDA examples
#define checkCudaErrors(val) check_cuda((val), #val, __FILE__, __LINE__)

void check_cuda(hipError_t result, char const *const func, const char *const file, int const line)
{
    if (result)
    {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " << file << ":" << line << " '"
                  << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

__device__ bool hit_sphere(const Vector3 &center, float radius, const Ray &r)
{
    Vector3 oc = r.origin() - center;
    float a = dot(r.direction(), r.direction());
    float b = 2.0f * dot(oc, r.direction());
    float c = dot(oc, oc) - radius * radius;
    float discriminant = b * b - 4.0f * a * c;
    return (discriminant > 0.0f);
}

__device__ Vector3 get_color(const Ray &r)
{
    if (hit_sphere(Vector3(0, 0, -1), 0.5, r))
    {
        return Vector3(1, 0, 0);
    }

    Vector3 unit_direction = unit_vector(r.direction());
    float t = 0.5f * (unit_direction.y() + 1.0f);
    return (1.0f - t) * Vector3(1.0, 1.0, 1.0) + t * Vector3(0.5, 0.7, 1.0);
}

__global__ void render(Vector3 *fb, int max_x, int max_y, Vector3 lower_left_corner, Vector3 horizontal,
                       Vector3 vertical, Vector3 origin)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= max_x) || (j >= max_y))
    {
        return;
    }

    int pixel_index = j * max_x + i;
    float u = float(i) / float(max_x);
    float v = float(j) / float(max_y);
    Ray r(origin, lower_left_corner + u * horizontal + v * vertical);
    fb[pixel_index] = get_color(r);
}

int main()
{
    int nx = 1200;
    int ny = 600;
    int tx = 8;
    int ty = 8;

    std::cerr << "Rendering a " << nx << "x" << ny << " image ";
    std::cerr << "in " << tx << "x" << ty << " blocks.\n";

    int num_pixels = nx * ny;
    size_t fb_size = num_pixels * sizeof(Vector3);

    // allocate FB
    Vector3 *fb;
    checkCudaErrors(hipMallocManaged((void **)&fb, fb_size));

    clock_t start, stop;
    start = clock();
    // Render our buffer
    dim3 blocks(nx / tx + 1, ny / ty + 1);
    dim3 threads(tx, ty);
    render<<<blocks, threads>>>(fb, nx, ny, Vector3(-2.0, -1.0, -1.0), Vector3(4.0, 0.0, 0.0), Vector3(0.0, 2.0, 0.0),
                                Vector3(0.0, 0.0, 0.0));
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    stop = clock();
    double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    std::cerr << "took " << timer_seconds << " seconds.\n";

    // Output FB as Image
    std::cout << "P3\n" << nx << " " << ny << "\n255\n";
    for (int j = ny - 1; j >= 0; j--)
    {
        for (int i = 0; i < nx; i++)
        {
            size_t pixel_index = j * nx + i;
            int ir = int(255.99 * fb[pixel_index].r());
            int ig = int(255.99 * fb[pixel_index].g());
            int ib = int(255.99 * fb[pixel_index].b());
            std::cout << ir << " " << ig << " " << ib << "\n";
        }
    }

    checkCudaErrors(hipFree(fb));
}